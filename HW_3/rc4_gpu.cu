#include <hip/hip_runtime.h>
#include <iostream>

#define TARGET_KEYSTREAM_SIZE 10
#define REPORT_INTERVAL 1000000

__device__ void rc4_key_schedule(unsigned char *key, int key_len, unsigned char *S) {
    int i, j = 0;
    for (i = 0; i < 256; i++) S[i] = i;
    for (i = 0; i < 256; i++) {
        j = (j + S[i] + key[i % key_len]) % 256;
        unsigned char temp = S[i];
        S[i] = S[j];
        S[j] = temp;
    }
}

__device__ void rc4_generate_keystream(unsigned char *S, int n, unsigned char *keystream) {
    int i = 0, j = 0;
    for (int t = 0; t < n; t++) {
        i = (i + 1) % 256;
        j = (j + S[i]) % 256;
        unsigned char temp = S[i];
        S[i] = S[j];
        S[j] = temp;
        keystream[t] = S[(S[i] + S[j]) % 256];
    }
}

__global__ void rc4_brute_force_kernel(unsigned char *target_keystream, unsigned char *key_found, int *found, int interval) {
    unsigned char key[5] = {80, 0, 0, 0, 0}; // Known first byte
    unsigned char S[256];
    unsigned char keystream[TARGET_KEYSTREAM_SIZE];

    // Calculate the key based on the thread ID
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    key[1] = (thread_id >> 16) & 0xFF;
    key[2] = (thread_id >> 8) & 0xFF;
    key[3] = thread_id & 0xFF;

    // Periodically print progress
    if (thread_id % interval == 0) {
        printf("Thread %d testing key: [%d, %d, %d, %d, %d]\n",
               thread_id, key[0], key[1], key[2], key[3], key[4]);
    }

    // Perform RC4 key scheduling and keystream generation
    rc4_key_schedule(key, 5, S);
    rc4_generate_keystream(S, TARGET_KEYSTREAM_SIZE, keystream);

    // Check if the keystream matches the target
    bool match = true;
    for (int i = 0; i < TARGET_KEYSTREAM_SIZE; i++) {
        if (keystream[i] != target_keystream[i]) {
            match = false;
            break;
        }
    }

    // If a match is found, store the key and set the found flag
    if (match && atomicExch(found, 1) == 0) {
        for (int i = 0; i < 5; i++) key_found[i] = key[i];
        printf("Thread %d found matching key: [%d, %d, %d, %d, %d]\n",
               thread_id, key[0], key[1], key[2], key[3], key[4]);
    }
}

int main() {
    unsigned char target_keystream[TARGET_KEYSTREAM_SIZE] = {130, 189, 254, 192, 238, 132, 216, 132, 82, 173};
    unsigned char *d_target_keystream, *d_key_found;
    int *d_found;
    unsigned char h_key_found[5] = {0};
    int h_found = 0;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_target_keystream, TARGET_KEYSTREAM_SIZE * sizeof(unsigned char));
    hipMalloc((void **)&d_key_found, 5 * sizeof(unsigned char));
    hipMalloc((void **)&d_found, sizeof(int));

    // Copy the target keystream to the GPU
    hipMemcpy(d_target_keystream, target_keystream, TARGET_KEYSTREAM_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_found, &h_found, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    int threads_per_block = 256;
    int num_blocks = (256 * 256 * 256 + threads_per_block - 1) / threads_per_block;
    std::cout << "Launching " << num_blocks << " blocks with " << threads_per_block << " threads per block." << std::endl;
    rc4_brute_force_kernel<<<num_blocks, threads_per_block>>>(d_target_keystream, d_key_found, d_found, REPORT_INTERVAL);

    // Copy the result back to the host
    hipMemcpy(&h_found, d_found, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_key_found, d_key_found, 5 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Check if a key was found
    if (h_found) {
        std::cout << "Key found: ";
        for (int i = 0; i < 5; i++) std::cout << (int)h_key_found[i] << " ";
        std::cout << std::endl;
    } else {
        std::cout << "No matching key found." << std::endl;
    }

    // Free GPU memory
    hipFree(d_target_keystream);
    hipFree(d_key_found);
    hipFree(d_found);

    return 0;
}
